#include <iostream>
#include <array>
#include <algorithm>
#include <numeric>

#include <hip/hip_runtime.h>

#include "hillclimb_cuda.h"
#include <thrust/copy.h>
#include <thrust/for_each.h>

cudaWrap::cudaWrap(bigram_table const& bgt, trigram_table const& tgt, std::vector<modalpha> const& ct)
{
	bgt_ = nullptr;
	tgt_ = nullptr;
	adt_ = nullptr;
	int deviceCount = 0;
	hipError_t err_id = hipGetDeviceCount(&deviceCount);
	if (err_id != hipSuccess)
	{
		std::cout << "hipGetDeviceCount returned " << err_id << "(" << hipGetErrorString(err_id) << '\n';
		return ;
	}
	if (deviceCount < 1)
	{
		std::cout << "No Cuda devices found\n";
		return ;
	}
	std::cout << "CUDA deviceCount = " << deviceCount << '\n';

	// copy in the bigram table of the day
	hipMalloc(reinterpret_cast<void**>(&bgt_), sizeof(bigram_table));
	hipMemcpy(bgt_, &bgt, sizeof(bigram_table), hipMemcpyHostToDevice);

	// copy in the trigram table of the day
	hipMalloc(reinterpret_cast<void**>(&tgt_), sizeof(trigram_table));
	hipMemcpy(tgt_, &tgt, sizeof(trigram_table), hipMemcpyHostToDevice);

	// reserve space for the arena
	hipMalloc(reinterpret_cast<void**>(&adt_), sizeof(arena_decode_t));

	// copy the ciphertext to the device
	ct_ = ct;
}

cudaWrap::~cudaWrap()
{
	hipFree(adt_);
	hipFree(tgt_);
	hipDeviceReset();
}

bool cudaWrap::cudaGood() const
{
	return tgt_ != nullptr && adt_ != nullptr;
}

void cudaWrap::set_arena(arena_decode_t const& a)
{
	// copy in the arena for this wheelorder/ring setting
	hipMemcpy(adt_, &a, sizeof(arena_decode_t), hipMemcpyHostToDevice);
}

void cudaWrap::sync_joblist_to_device(std::vector<cudaJob> const& jl)
{
	vjd_ = vjh_;
}

void cudaWrap::sync_joblist_from_device(std::vector<cudaJob>& jl)
{
	vjh_ = vjd_;
}

// localise the hillclimb code(s) to here for now as functors
//
class fast_decoder_ref
{
private:
	arena_decode_t::pos_t const*	ai_;
	thrust::device_vector<modalpha> vo_;

public:
	fast_decoder_ref() = delete;
	__device__
	fast_decoder_ref(arena_decode_t::pos_t const* ai) : ai_(ai)
	{
	}
	__device__
	thrust::device_vector<modalpha> const& decode(thrust::device_vector<modalpha> const& ct, stecker const& s)
	{
		vo_.resize(ct.size());
		std::transform(ct.begin(), ct.end(), ai_, vo_.begin(), [&](auto c, auto const& a)
			{
				// in stecker
				auto o = s.Eval(c);
				// rotor cache
				o = a[o.Val_()];
				// out stecker 
				o = s.Eval(o);
				return o;
			});

		return vo_;
	}
};
#if 0
double index_of_coincidence(thrust::device_vector<modalpha> const& pt)
{
	std::array<unsigned, alpha_max> tab;
	tab.fill(0);

	// count
	std::for_each(pt.begin(), pt.end(), [&tab](auto c)
		{
			++tab[c.Val()];
		});
	// calculate
	double nn = double(pt.size()) * double(pt.size() - 1);

	return 	std::inner_product(std::begin(tab), std::end(tab), 0.0, std::plus<double>{}, [nn](auto n)
		{
			return double(n * (n - 1)) / nn;
		});
}
#endif
struct bigram_score_op
{
	bigram_table const* bgt_;
	__device__
	bigram_score_op(bigram_table const* bgt) : bgt_(bgt)
	{}
	__device__
	unsigned operator()(thrust::device_vector<modalpha> const& pt)
	{
		unsigned score = 0;
		auto b = pt.begin();
		auto n = b + 1;
		while (n != pt.end())
		{
			score += bgt_->wt(*b, *n);
			++b;
			++n;
		}
		// normalise to message length here, right down at the source.
		return score / (static_cast<unsigned>(pt.size()) - 1);
	}
};

__device__ struct trigram_score_op
{
	trigram_table const* tgt_;
	__device__
	trigram_score_op(trigram_table const* tgt) : tgt_(tgt)
	{}
	__device__
	unsigned operator()(thrust::device_vector<modalpha> const& pt)
	{
		unsigned score = 0;
		auto b = pt.begin();
		auto m = b + 1;
		auto n = m + 1;
		while (n != pt.end())
		{
			score += tgt_->wt(*b, *m, *n);
			++b;
			++m;
			++n;
		}
		return score / (static_cast<unsigned>(pt.size()) - 2);
	}
};

template<typename F, typename FD, size_t max_stecker = 10 > auto hillclimb_base_fast(thrust::device_vector<modalpha> const& ct, F eval_fn, double iocb, FD& fd, stecker& s_base)
{
	stecker s = s_base;
	stecker s_b;
	auto vo = fd.decode(ct, s);
#if 0
	auto iocs = index_of_coincidence(vo);
	if (iocs * .95 < iocb)
		return 0U;
#endif
	// establish the baseline
	auto scr = eval_fn(vo);
	bool improved = true;
	while (improved)
	{
		improved = false;
		modalpha mx = 0;
		modalpha my = 0;
		for (int fi = 0; fi < alpha_max; ++fi)
		{
			modalpha f{ fi };
			for (int ti = fi; ti < alpha_max; ++ti)
			{
				modalpha t{ ti };
				s_b = s;
				s.Apply(f, t);
				vo = fd.decode(ct, s);
				auto scrn = eval_fn(vo);
				if (scrn > scr && s.Count() < max_stecker + 1)
				{
					mx = f;
					my = t;
					scr = scrn;
					improved = true;
				}
				s = s_b;
			}
		}
		if (improved)
			s.Apply(mx, my);
	}
	s_base = s;

	return scr;
}

__device__ struct hillclimb_bgtg_fast
{
	thrust::device_vector<modalpha> const& ct_;
	arena_decode_t const* ai_;
	bigram_table const* bgt_;
	trigram_table const* tgt_;

	__device__
	hillclimb_bgtg_fast(thrust::device_vector<modalpha> const& ct, arena_decode_t const* ai, bigram_table const* bgt, trigram_table const* tgt) : ct_(ct), ai_(ai), bgt_(bgt), tgt_(tgt)
	{}

	__device__
	void operator()( cudaJob& cj)
	{
		fast_decoder_ref fd(ai_->arena_.data() + cj.off_);
		hillclimb_base_fast(ct_, bigram_score_op(bgt_), 0.0, fd, cj.s_);
		cj.scr_ = hillclimb_base_fast(ct_, trigram_score_op(tgt_), 0.0, fd, cj.s_);
	}
};

void cudaWrap::proc()
{
	thrust::for_each(vjd_.begin(), vjd_.end(), hillclimb_bgtg_fast(ct_, adt_, bgt_, tgt_));
}