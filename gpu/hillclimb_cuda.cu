#include "hip/hip_runtime.h"
#include <iostream>
#include <array>
#include <algorithm>
#include <numeric>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "hillclimb_cuda.h"

cudaWrap::cudaWrap(bigram_table const& bgt, trigram_table const& tgt, std::vector<modalpha> const& ct)
{
	bgt_ = nullptr;
	tgt_ = nullptr;
	adt_ = nullptr;
	ct_ = nullptr;
	jl_ = nullptr;
	int deviceCount = 0;
	hipError_t err_id = hipGetDeviceCount(&deviceCount);
	if (err_id != hipSuccess)
	{
		std::cout << "hipGetDeviceCount returned " << err_id << "(" << hipGetErrorString(err_id) << '\n';
		return ;
	}
	if (deviceCount < 1)
	{
		std::cout << "No Cuda devices found\n";
		return ;
	}
	std::cout << "CUDA deviceCount = " << deviceCount << '\n';

	// copy in the bigram table of the day
	hipMalloc(reinterpret_cast<void**>(&bgt_), sizeof(bigram_table));
	hipMemcpy(bgt_, &bgt, sizeof(bigram_table), hipMemcpyHostToDevice);

	// copy in the trigram table of the day
	hipMalloc(reinterpret_cast<void**>(&tgt_), sizeof(trigram_table));
	hipMemcpy(tgt_, &tgt, sizeof(trigram_table), hipMemcpyHostToDevice);

	// reserve space for the arena
	hipMalloc(reinterpret_cast<void**>(&adt_), sizeof(arena_decode_t));

	// reserve and copy ciphertext
	ctl_ = static_cast<unsigned>(ct.size());
	hipMalloc(reinterpret_cast<void**>(&ct_), ctl_);
	hipMemcpy(ct_, ct.data(), ctl_, hipMemcpyHostToDevice);
}

cudaWrap::~cudaWrap()
{
	hipFree(adt_);
	hipFree(tgt_);
	hipFree(ct_);
	hipFree(jl_);
	hipDeviceReset();
}

bool cudaWrap::cudaGood() const
{
	if (tgt_ != nullptr && adt_ != nullptr && ct_ != nullptr)
	{
		// wait
		auto err = hipDeviceSynchronize();
		if (err != hipSuccess)
		{
			std::cout << "Cuda Launch Error - " << err << ": " << hipGetErrorString(err) << '\n';
			return false;
		}
		return true;
	}
	return false;
}

void cudaWrap::set_arena(arena_decode_t const& a)
{
	// copy in the arena for this wheelorder/ring setting
	hipMemcpy(adt_, &a, sizeof(arena_decode_t), hipMemcpyHostToDevice);
}

void cudaWrap::sync_joblist_to_device(std::vector<cudaJob> const& jl)
{
	if (jl_ == nullptr)
	{
		jls_ = static_cast<unsigned>(jl.size());
		auto err = hipMalloc(reinterpret_cast<void**>(&jl_), sizeof(cudaJob) * jls_);
		if (err != hipSuccess)
		{
			std::cout << "hipMalloc Error - " << err << ": " << hipGetErrorString(err) << '\n';
		}

	}
	auto err = hipMemcpy(jl_, jl.data(), jls_ * sizeof(cudaJob), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		std::cout << "hipMemcpy Error - " << err << ": " << hipGetErrorString(err) << '\n';
	}
}

void cudaWrap::sync_joblist_from_device(std::vector<cudaJob>& jl)
{
	auto err = hipMemcpy(jl.data(), jl_, jls_ * sizeof(cudaJob), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		std::cout << "cudaMemcpy2 Error - " << err << ": " << hipGetErrorString(err) << '\n';
	}
}

// localise the hillclimb code(s) to here for now as functors
//
class fast_decoder_ptr
{
private:
	modalpha const*	ai_;
	modalpha		vo_[256];
public:
	fast_decoder_ptr() = delete;
	__device__
	fast_decoder_ptr(modalpha const* ai) : ai_(ai)
	{
	}
	__device__
	~fast_decoder_ptr()
	{
	}
	__device__
	modalpha* decode(modalpha const* ctb, modalpha const* cte, stecker const& s)
	{
		modalpha const* a = ai_;
		auto pt = vo_;
		while (ctb != cte)
		{
			auto o = s.Eval(*ctb);
			o = a[o.Val()];
			o = s.Eval(o);
			*pt = o;
			++pt;
			++ctb;
			a += alpha_max;
		}
		return vo_;
	}
};

__device__ double index_of_coincidence(modalpha const* ctb, unsigned ctl)
{
	unsigned tab[alpha_max];
	for (auto& t : tab)
		t = 0;

	// count
	while(ctb != ctb + ctl)
		++tab[(*ctb).Val()];

	// calculate
	double nn = double(ctl) * double(ctl - 1);
	double rv = 0.0;
	for (auto t : tab)
		rv += double(t * (t - 1)) / nn;
	return rv;
}

struct bigram_score_op
{
	bigram_table const* bgt_;
	__device__
	bigram_score_op(bigram_table const* bgt) : bgt_(bgt)
	{}
	__device__
	unsigned operator()(modalpha const* ptb, modalpha const* pte)
	{
		unsigned score = 0;
		auto b = ptb;
		auto n = ptb + 1;
		while (n != pte)
		{
			score += bgt_->wt(*b, *n);
			++b;
			++n;
		}
		// normalise to message length here, right down at the source.
		return score / (static_cast<unsigned>(pte - ptb) - 1);
	}
};

struct trigram_score_op
{
	trigram_table const* tgt_;
	__device__
	trigram_score_op(trigram_table const* tgt) : tgt_(tgt)
	{}
	__device__
	unsigned operator()(modalpha const* ptb, modalpha const* pte)
	{
		unsigned score = 0;
		auto b = ptb;
		auto m = b + 1;
		auto n = m + 1;
		while (n != pte)
		{
			score += tgt_->wt(*b, *m, *n);
			++b;
			++m;
			++n;
		}
		return score / (static_cast<unsigned>(pte - ptb) - 2);
	}
};


template<typename F, typename FD, size_t max_stecker = 10 > __device__ unsigned hillclimb_base_fast(modalpha const* ctb, modalpha const* cte, F eval_fn, double iocb, FD& fd, stecker* s_base)
{
	stecker s = *s_base;
	stecker s_b;
	unsigned ctl = cte - ctb;
	auto vo = fd.decode(ctb, cte, s);
	auto iocs = index_of_coincidence(vo, ctl);
	if (iocs * .95 < iocb)
		return 0U;
	// establish the baseline
	auto scr = eval_fn(vo, vo + ctl);
	while (1)
	{
		bool improved = false;
		modalpha mx = 0;
		modalpha my = 0;
		for (int fi = 0; fi < alpha_max; ++fi)
		{
			modalpha f{ fi };
			for (int ti = fi; ti < alpha_max; ++ti)
			{
				modalpha t{ ti };
				s_b = s;
				s.Apply(f, t);
				vo = fd.decode(ctb, cte, s);
				auto scrn = eval_fn(vo, vo + ctl);
				if (scrn > scr /*&& s.Count() < max_stecker + 1*/)
				{
					mx = f;
					my = t;
					scr = scrn;
					improved = true;
				}
				s = s_b;
			}
		}
		if (!improved)
			break;
		s.Apply(mx, my);
	}
	*s_base = s;
	return scr;
}

// the simple form of hillclimb, best of bg followed by best of tg
__device__ void hillclimb_bgtg_fast(modalpha const* ctb, modalpha const* cte, arena_decode_t const* ai, bigram_table const* bgt, trigram_table const* tgt, cudaJob& cj)
{
	fast_decoder_ptr fd(ai->arena_ + cj.off_ * alpha_max);
	hillclimb_base_fast(ctb, cte, bigram_score_op(bgt), 0.0, fd, &cj.s_);
	cj.scr_ = hillclimb_base_fast(ctb, cte, trigram_score_op(tgt), 0.0, fd, &cj.s_);
}

__global__ void process_hillclimb(cudaJob* jl, unsigned jls, modalpha* ct, unsigned ctl, arena_decode_t* ai, bigram_table* bgt, trigram_table* tgt)
{
	// figure out which cudaJob refers and call the actual worker fn.
	unsigned j = blockIdx.x * blockDim.x + threadIdx.x;
	if (j >= jls)
		return;
	hillclimb_bgtg_fast(ct, ct + ctl, ai, bgt, tgt, *(jl + j));
}
#if 0
// try partial exhaustion of the combinations
//
// use the fast decoder
__device__ void hillclimb_partial_exhaust2_fast(modalpha const* ctb, modalpha const* cte, arena_decode_t const* ai, trigram_table const* tgt, cudaJob& cj)
{
	const modalpha f1 = alpha::E;
	const modalpha f2 = alpha::N;
	fast_decoder_ptr fd(ai->arena_ + cj.off_ * alpha_max);
	stecker s_b = cj.s_;
	stecker s_best;
	// establish the baseline
	unsigned ctl = cte - ctb;
	auto vo = fd.decode(ctb, cte, s_b);
	auto ef = trigram_score_op(tgt);
	auto scr = ef(vo, vo + ctl);
	auto iocb = index_of_coincidence(vo, ctl);
	for (int ti1 = 0; ti1 < alpha_max; ++ti1)
	{
		modalpha t1{ ti1 };
		if (ti1 == f2)
			continue;
		for (int ti2 = 0; ti2 < alpha_max; ++ti2)
		{
			modalpha t2{ ti2 };
			if (t2 == t1 || t2 == f1)
				continue;
			s_b = cj.s_;
			s_b.Apply(f2, t2);
			s_b.Apply(f1, t1);
			auto scrn = hillclimb_base_fast(ctb, cte, ef, iocb, fd, &s_b);
			if (scrn > scr)
			{
				s_best = s_b;
				scr = scrn;
			}
		}
	}
	cj.s_ = s_best;
	cj.scr_ = scr;
}

__global__ void process_hillclimb_ex(cudaJob* jl, unsigned jls, modalpha* ct, unsigned ctl, arena_decode_t* ai, trigram_table* tgt)
{
	// figure out which cudaJob refers and call the actual worker fn.
	unsigned j = blockIdx.x * blockDim.x + threadIdx.x;
	if (j >= jls)
		return;
	hillclimb_partial_exhaust2_fast(ct, ct + ctl, ai, tgt, *(jl + j));
}
#endif
void cudaWrap::run_gpu_process()
{
	// assume (for now) 32 threads per warp and so threads per block 
	// is cj size / 32.
	unsigned tpb = (jls_ + 31) / 32;
	// start
	dim3 block(tpb);
	dim3 grid(32);
#if 0
	unsigned step = jls_ / 16;
	unsigned off = 0;
	while (off < jls_)
	{
		process_hillclimb << <grid, block >> > (jl_ + off, step, ct_, ctl_, adt_, bgt_, tgt_);
		auto err = hipDeviceSynchronize();	if (err != hipSuccess)
		{
			std::cout << "hipDeviceSynchronize Error - " << err << ": " << hipGetErrorString(err) << '\n';
			return;
		}
		off += step;
	}
#else
	process_hillclimb <<<grid, block>>> (jl_, jls_, ct_, ctl_, adt_, bgt_, tgt_);
#endif
}

void cudaWrap::run_gpu_process_ex()
{
#if 0
	// assume (for now) 32 threads per warp and so threads per block 
	// is cj size / 32.
	unsigned tpb = (jls_ + 31) / 32;
	// start
	dim3 block(tpb);
	dim3 grid(32);
	process_hillclimb_ex <<<grid, block>>> (jl_, jls_, ct_, ctl_, adt_, tgt_);
#endif
}
