#include "hip/hip_runtime.h"
#include <iostream>
#include <array>
#include <algorithm>
#include <numeric>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "hillclimb_cuda.h"

cudaWrap::cudaWrap(bigram_table const& bgt, trigram_table const& tgt, std::vector<modalpha> const& ct)
{
	bgt_ = nullptr;
	tgt_ = nullptr;
	adt_ = nullptr;
	ct_ = nullptr;
	jl_ = nullptr;
	int deviceCount = 0;
	hipError_t err_id = hipGetDeviceCount(&deviceCount);
	if (err_id != hipSuccess)
	{
		std::cout << "hipGetDeviceCount returned " << err_id << "(" << hipGetErrorString(err_id) << '\n';
		return ;
	}
	if (deviceCount < 1)
	{
		std::cout << "No Cuda devices found\n";
		return ;
	}
	std::cout << "CUDA deviceCount = " << deviceCount << '\n';

	// copy in the bigram table of the day
	hipMalloc(reinterpret_cast<void**>(&bgt_), sizeof(bigram_table));
	hipMemcpy(bgt_, &bgt, sizeof(bigram_table), hipMemcpyHostToDevice);

	// copy in the trigram table of the day
	hipMalloc(reinterpret_cast<void**>(&tgt_), sizeof(trigram_table));
	hipMemcpy(tgt_, &tgt, sizeof(trigram_table), hipMemcpyHostToDevice);

	// reserve space for the arena
	hipMalloc(reinterpret_cast<void**>(&adt_), sizeof(arena_decode_t));

	// reserve and copy ciphertext
	ctl_ = static_cast<unsigned>(ct.size());
	hipMalloc(reinterpret_cast<void**>(&ct_), ctl_);
	hipMemcpy(ct_, ct.data(), ctl_, hipMemcpyHostToDevice);
}

cudaWrap::~cudaWrap()
{
	hipFree(adt_);
	hipFree(tgt_);
	hipFree(ct_);
	hipFree(jl_);
	hipDeviceReset();
}

bool cudaWrap::cudaGood() const
{
	if (tgt_ != nullptr && adt_ != nullptr && ct_ != nullptr)
	{
		// wait
		auto err = hipDeviceSynchronize();
		if (err != hipSuccess)
		{
			std::cout << "Cuda Launch Error - " << err << ": " << hipGetErrorString(err) << '\n';
			return false;
		}
	}
	return true;
}

void cudaWrap::set_arena(arena_decode_t const& a)
{
	// copy in the arena for this wheelorder/ring setting
	hipMemcpy(adt_, &a, sizeof(arena_decode_t), hipMemcpyHostToDevice);
}

void cudaWrap::sync_joblist_to_device(std::vector<cudaJob> const& jl)
{
	if (jl_ == nullptr)
	{
		jls_ = static_cast<unsigned>(jl.size());
		auto err = hipMalloc(reinterpret_cast<void**>(&jl_), sizeof(cudaJob) * jls_);
		if (err != hipSuccess)
		{
			std::cout << "hipMalloc Error - " << err << ": " << hipGetErrorString(err) << '\n';
		}

	}
	auto err = hipMemcpy(jl_, jl.data(), jls_ * sizeof(cudaJob), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		std::cout << "hipMemcpy Error - " << err << ": " << hipGetErrorString(err) << '\n';
	}
}

void cudaWrap::sync_joblist_from_device(std::vector<cudaJob>& jl)
{
	auto err = hipMemcpy(jl.data(), jl_, jls_ * sizeof(cudaJob), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		std::cout << "cudaMemcpy2 Error - " << err << ": " << hipGetErrorString(err) << '\n';
	}
}

// localise the hillclimb code(s) to here for now as functors
//
class fast_decoder_ptr
{
private:
	modalpha const*	ai_;
	modalpha		vo_[256];
public:
	fast_decoder_ptr() = delete;
	__device__
	fast_decoder_ptr(modalpha const* ai) : ai_(ai)
	{
	}
	__device__
	~fast_decoder_ptr()
	{
	}
	__device__
	modalpha* decode(modalpha const* ctb, modalpha const* cte, stecker const& s)
	{
		modalpha const* a = ai_;
		auto pt = vo_;
		while (ctb != cte)
		{
			auto o = s.Eval(*ctb);
			o = a[o.Val()];
			o = s.Eval(o);
			*pt = o;
			++pt;
			++ctb;
			a += alpha_max;
		}
		return vo_;
	}
};
#if 0
double index_of_coincidence(thrust::device_vector<modalpha> const& pt)
{
	std::array<unsigned, alpha_max> tab;
	tab.fill(0);

	// count
	std::for_each(pt.begin(), pt.end(), [&tab](auto c)
		{
			++tab[c.Val()];
		});
	// calculate
	double nn = double(pt.size()) * double(pt.size() - 1);

	return 	std::inner_product(std::begin(tab), std::end(tab), 0.0, std::plus<double>{}, [nn](auto n)
		{
			return double(n * (n - 1)) / nn;
		});
}
#endif
struct bigram_score_op
{
	bigram_table const* bgt_;
	__device__
	bigram_score_op(bigram_table const* bgt) : bgt_(bgt)
	{}
	__device__
	unsigned operator()(modalpha const* ptb, modalpha const* pte)
	{
		unsigned score = 0;
		auto b = ptb;
		auto n = ptb + 1;
		while (n != pte)
		{
			score += bgt_->wt(*b, *n);
			++b;
			++n;
		}
		// normalise to message length here, right down at the source.
		return score / (static_cast<unsigned>(pte - ptb) - 1);
	}
};

struct trigram_score_op
{
	trigram_table const* tgt_;
	__device__
	trigram_score_op(trigram_table const* tgt) : tgt_(tgt)
	{}
	__device__
	unsigned operator()(modalpha const* ptb, modalpha const* pte)
	{
		unsigned score = 0;
		auto b = ptb;
		auto m = b + 1;
		auto n = m + 1;
		while (n != pte)
		{
			score += tgt_->wt(*b, *m, *n);
			++b;
			++m;
			++n;
		}
		return score / (static_cast<unsigned>(pte - ptb) - 2);
	}
};


template<typename F, typename FD, size_t max_stecker = 10 > __device__ unsigned hillclimb_base_fast(modalpha const* ctb, modalpha const* cte, F eval_fn, double iocb, FD& fd, stecker* s_base)
{
	stecker s = *s_base;
	stecker s_b;
	unsigned ctl = cte - ctb;
	auto vo = fd.decode(ctb, cte, s);
#if 0
	auto iocs = index_of_coincidence(vo);
	if (iocs * .95 < iocb)
		return 0U;
#endif
	// establish the baseline
	auto scr = eval_fn(vo, vo + ctl);
	while (1)
	{
		bool improved = false;
		modalpha mx = 0;
		modalpha my = 0;
		for (int fi = 0; fi < alpha_max; ++fi)
		{
			modalpha f{ fi };
			for (int ti = fi; ti < alpha_max; ++ti)
			{
				modalpha t{ ti };
				s_b = s;
				s.Apply(f, t);
				vo = fd.decode(ctb, cte, s);
				auto scrn = eval_fn(vo, vo + ctl);
				if (scrn > scr /*&& s.Count() < max_stecker + 1*/)
				{
					mx = f;
					my = t;
					scr = scrn;
					improved = true;
				}
				s = s_b;
			}
		}
		if (!improved)
			break;
		s.Apply(mx, my);
	}
	*s_base = s;
	return scr;
}

__device__ void hillclimb_bgtg_fast(modalpha const* ctb, modalpha const* cte, arena_decode_t const* ai, bigram_table const* bgt, trigram_table const* tgt, cudaJob& cj, unsigned k)
{
	fast_decoder_ptr fd(ai->arena_ + cj.off_ * alpha_max);
	hillclimb_base_fast(ctb, cte, bigram_score_op(bgt), 0.0, fd, &cj.s_);
	cj.scr_ = hillclimb_base_fast(ctb, cte, trigram_score_op(tgt), 0.0, fd, &cj.s_);
}

__global__ void process_hillclimb(cudaJob* jl, unsigned jls, modalpha* ct, unsigned ctl, arena_decode_t* ai, bigram_table* bgt, trigram_table* tgt)
{
	// figure out which cudaJob refers and call the actual worker fn.
	unsigned j = blockIdx.x * blockDim.x + threadIdx.x;
	if (j >= jls)
	{
//		printf("filter ob j = %d (%d, %d, %d) (%d, %d, %d) (%d, %d, %d)\n", j, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, threadIdx.x, threadIdx.y, threadIdx.z);
		return;
	}
	hillclimb_bgtg_fast(ct, ct + ctl, ai, bgt, tgt, *(jl + j), j);
}

void cudaWrap::run_gpu_process()
{
	// assume (for now) 32 threads per warp and so threads per block 
	// is cj size / 32.
	unsigned tpb = (jls_ + 31) / 32;
	// start
	dim3 block(tpb);
	dim3 grid(32);
	process_hillclimb <<<grid, block>>> (jl_, jls_, ct_, ctl_, adt_, bgt_, tgt_);
}