#include <iostream>
#include <array>

#include <hip/hip_runtime.h>

#include "hillclimb_cuda.h"
#include <thrust/copy.h>
#include <thrust/for_each.h>

cudaWrap::cudaWrap(trigram_table const& tgt, std::vector<modalpha> const& ct)
{
	tgt_ = nullptr;
	adt_ = nullptr;
	int deviceCount = 0;
	hipError_t err_id = hipGetDeviceCount(&deviceCount);
	if (err_id != hipSuccess)
	{
		std::cout << "hipGetDeviceCount returned " << err_id << "(" << hipGetErrorString(err_id) << '\n';
		return ;
	}
	if (deviceCount < 1)
	{
		std::cout << "No Cuda devices found\n";
		return ;
	}
	std::cout << "CUDA deviceCount = " << deviceCount << '\n';

	// copy in the trigram table of the day
	hipMalloc(&tgt_, sizeof(trigram_table));
	hipMemcpy(tgt_, &tgt, sizeof(trigram_table), hipMemcpyHostToDevice);

	// reserve space for the arena
	hipMalloc(&adt_, sizeof(arena_decode_t));

	// copy in the ciphertext
	thrust::host_vector<modalpha> cth(ct.size());
	thrust::copy(ct.begin(), ct.end(), cth.begin());
	ct_ = cth;
}

cudaWrap::~cudaWrap()
{
	hipFree(adt_);
	hipFree(tgt_);
	hipDeviceReset();
}

bool cudaWrap::cudaGood() const
{
	return tgt_ != nullptr && adt_ != nullptr;
}

void cudaWrap::set_arena(arena_decode_t const& a)
{
	// copy in the arena for this wheelorder/ring setting
	hipMemcpy(adt_, &a, sizeof(arena_decode_t), hipMemcpyHostToDevice);
}

#if 0
trigram_table*  tgt_;
arena_decode_t* arena_;

bool init_cuda(trigram_table const& tgt)
{
	tgt_ = nullptr;
	arena_ = nullptr;
	int deviceCount = 0;
	hipError_t err_id = hipGetDeviceCount(&deviceCount);
	if (err_id != hipSuccess)
	{
		std::cout << "hipGetDeviceCount returned " << err_id << "(" << hipGetErrorString(err_id) << '\n';
		return false;
	}
	if (deviceCount < 1)
	{
		std::cout << "No Cuda devices found\n";
		return false;
	}
	std::cout << "CUDA deviceCount = " << deviceCount << '\n';

	// copy in the trigram table of the day
	hipMalloc(&tgt_, sizeof(trigram_table));
	hipMemcpy(tgt_, &tgt, sizeof(trigram_table), hipMemcpyHostToDevice);

	// reserve space for the arena
	hipMalloc(&arena_, sizeof(arena_decode_t));

	return false;
}

void set_arena(arena_decode_t const& arena)
{
	// copy in the arena for this wheelorder/ring setting
	hipMemcpy(arena_, &arena, sizeof(arena), hipMemcpyHostToDevice);
}

void uninit_cuda()
{
	hipFree(arena_);
	hipFree(tgt_);
	hipDeviceReset();
}
#endif