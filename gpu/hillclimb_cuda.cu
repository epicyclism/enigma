#include <iostream>
#include <array>
#include <algorithm>
#include <numeric>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "hillclimb_cuda.h"

cudaWrap::cudaWrap(bigram_table const& bgt, trigram_table const& tgt, std::vector<modalpha> const& ct)
{
	bgt_ = nullptr;
	tgt_ = nullptr;
	adt_ = nullptr;
	ct_ = nullptr;
	jl_ = nullptr;
	int deviceCount = 0;
	hipError_t err_id = hipGetDeviceCount(&deviceCount);
	if (err_id != hipSuccess)
	{
		std::cout << "hipGetDeviceCount returned " << err_id << "(" << hipGetErrorString(err_id) << '\n';
		return ;
	}
	if (deviceCount < 1)
	{
		std::cout << "No Cuda devices found\n";
		return ;
	}
	std::cout << "CUDA deviceCount = " << deviceCount << '\n';

	// copy in the bigram table of the day
	hipMalloc(reinterpret_cast<void**>(&bgt_), sizeof(bigram_table));
	hipMemcpy(bgt_, &bgt, sizeof(bigram_table), hipMemcpyHostToDevice);

	// copy in the trigram table of the day
	hipMalloc(reinterpret_cast<void**>(&tgt_), sizeof(trigram_table));
	hipMemcpy(tgt_, &tgt, sizeof(trigram_table), hipMemcpyHostToDevice);

	// reserve space for the arena
	hipMalloc(reinterpret_cast<void**>(&adt_), sizeof(arena_decode_t));

	// reserve and copy ciphertext
	ctl_ = static_cast<unsigned>(ct.size());
	hipMalloc(reinterpret_cast<void**>(&ct_), ctl_);
	hipMemcpy(ct_, ct.data(), ctl_, hipMemcpyHostToDevice);
}

cudaWrap::~cudaWrap()
{
	hipFree(adt_);
	hipFree(tgt_);
	hipFree(ct_);
	hipFree(jl_);
	hipDeviceReset();
}

bool cudaWrap::cudaGood() const
{
	return tgt_ != nullptr && adt_ != nullptr && ct_ != nullptr ;
}

void cudaWrap::set_arena(arena_decode_t const& a)
{
	// copy in the arena for this wheelorder/ring setting
	hipMemcpy(adt_, &a, sizeof(arena_decode_t), hipMemcpyHostToDevice);
}

void cudaWrap::sync_joblist_to_device(std::vector<cudaJob> const& jl)
{
	if (jl_ == nullptr)
	{
		jls_ = static_cast<unsigned>(jl.size());
		hipMalloc(reinterpret_cast<void**>(&jl_), sizeof(cudaJob) * jls_);
	}
	hipMemcpy(jl_, jl.data(), jls_ * sizeof(cudaJob), hipMemcpyHostToDevice);
}

void cudaWrap::sync_joblist_from_device(std::vector<cudaJob>& jl)
{
	hipMemcpy(jl.data(), jl_, jls_ * sizeof(cudaJob), hipMemcpyDeviceToHost);
}

// localise the hillclimb code(s) to here for now as functors
//
class fast_decoder_ptr
{
private:
	modalpha const*	ai_;
	modalpha		vo_[256];
public:
	fast_decoder_ptr() = delete;
	__device__
	fast_decoder_ptr(modalpha const* ai) : ai_(ai)
	{
	}
	__device__
	~fast_decoder_ptr()
	{
	}
	__device__
	modalpha* decode(modalpha const* ctb, modalpha const* cte, stecker const& s)
	{
		modalpha const* a = ai_;
		auto pt = vo_;
		while (ctb != cte)
		{
			auto o = s.Eval(*ctb);
			o = a[o.Val()];
			o = s.Eval(o);
			*pt = o;
			++pt;
			++ctb;
			a += alpha_max;
		}
		return vo_;
	}
};
#if 0
double index_of_coincidence(thrust::device_vector<modalpha> const& pt)
{
	std::array<unsigned, alpha_max> tab;
	tab.fill(0);

	// count
	std::for_each(pt.begin(), pt.end(), [&tab](auto c)
		{
			++tab[c.Val()];
		});
	// calculate
	double nn = double(pt.size()) * double(pt.size() - 1);

	return 	std::inner_product(std::begin(tab), std::end(tab), 0.0, std::plus<double>{}, [nn](auto n)
		{
			return double(n * (n - 1)) / nn;
		});
}
#endif
struct bigram_score_op
{
	bigram_table const* bgt_;
	__device__
	bigram_score_op(bigram_table const* bgt) : bgt_(bgt)
	{}
	__device__
	unsigned operator()(modalpha const* ptb, modalpha const* pte)
	{
		unsigned score = 0;
		auto b = ptb;
		auto n = ptb + 1;
		while (n != pte)
		{
			score += bgt_->wt(*b, *n);
			++b;
			++n;
		}
		// normalise to message length here, right down at the source.
		return score / (static_cast<unsigned>(pte - ptb) - 1);
	}
};

struct trigram_score_op
{
	trigram_table const* tgt_;
	__device__
	trigram_score_op(trigram_table const* tgt) : tgt_(tgt)
	{}
	__device__
	unsigned operator()(modalpha const* ptb, modalpha const* pte)
	{
		unsigned score = 0;
		auto b = ptb;
		auto m = b + 1;
		auto n = m + 1;
		while (n != pte)
		{
			score += tgt_->wt(*b, *m, *n);
			++b;
			++m;
			++n;
		}
		return score / (static_cast<unsigned>(pte - ptb) - 2);
	}
};


template<typename F, typename FD, size_t max_stecker = 10 > __device__ unsigned hillclimb_base_fast(modalpha const* ctb, modalpha const* cte, F eval_fn, double iocb, FD& fd, stecker* s_base)
{
	stecker s = *s_base;
	stecker s_b;
	unsigned ctl = cte - ctb;
	auto vo = fd.decode(ctb, cte, s);
#if 0
	auto iocs = index_of_coincidence(vo);
	if (iocs * .95 < iocb)
		return 0U;
#endif
	// establish the baseline
	auto scr = eval_fn(vo, vo + ctl);
	bool improved = true;
	while (improved)
	{
		improved = false;
		modalpha mx = 0;
		modalpha my = 0;
		for (int fi = 0; fi < alpha_max; ++fi)
		{
			modalpha f{ fi };
			for (int ti = fi; ti < alpha_max; ++ti)
			{
				modalpha t{ ti };
				s_b = s;
				s.Apply(f, t);
				vo = fd.decode(ctb, cte, s);
				auto scrn = eval_fn(vo, vo + ctl);
				if (scrn > scr && s.Count() < max_stecker + 1)
				{
					mx = f;
					my = t;
					scr = scrn;
					improved = true;
				}
				s = s_b;
			}
		}
		if (improved)
			s.Apply(mx, my);
	}
	*s_base = s;

	return scr;
}

__device__ void hillclimb_bgtg_fast(modalpha const* ctb, modalpha const* cte, arena_decode_t const* ai, bigram_table const* bgt, trigram_table const* tgt, cudaJob& cj)
{
	fast_decoder_ptr fd(ai->arena_ + cj.off_ * alpha_max);
	hillclimb_base_fast(ctb, cte, bigram_score_op(bgt), 0.0, fd, &cj.s_);
	cj.scr_ = hillclimb_base_fast(ctb, cte, trigram_score_op(tgt), 0.0, fd, &cj.s_);
}

void cudaWrap::proc()
{
}